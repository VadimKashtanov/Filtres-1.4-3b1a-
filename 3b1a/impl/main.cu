#include "hip/hip_runtime.h"
#include "main.cuh"

#include "../impl_tmpl/tmpl_etc.cu"

static void plume_pred(Mdl_t * mdl, uint t0, uint t1) {
	float * ancien = mdl_pred(mdl, t0, t1, 3);
	printf("PRED GENERALE = ");
	FOR(0, p, P) printf(" %f%% ", 100*ancien[p]);
	printf("\n");
	free(ancien);
};

float pourcent_masque_nulle[C] = {0};

float * pourcent_masque = de_a(0.0, 0.00, C);

//	! A FAIRE ! :
//		selection (mutation de +/- 1 ligne (de meme source))
//

float * alpha = de_a(2e-3, 2e-3, C);

//	## (x/3) * (x-2)**2                     ##
//	## score(x) + rnd()*abs(score(x))*0.05  ##

PAS_OPTIMISER()
int main(int argc, char ** argv) {
	MSG("S(x) Ajouter un peut d'aléatoire");
	MSG("S(x) Eventuellement faire des prediction plus lointaines");
	//	-- Init --
	srand(0);
	hipSetDevice(0);
	titre(" Charger tout ");   charger_tout();

	//	-- Verification --
	//titre("Verifier MDL");     verif_mdl_1e5();

	//===============
	titre("  Programme Generale  ");

	/*uint Y[C] = {
		512,
		256,256,
		128,128,128,
		64,
		32,
		16,
		8,
		4,
		P
	};
	uint insts[C] = {
		FILTRES_PRIXS,
		DOT1D,DOT1D,
		DOT1D,DOT1D,
		DOT1D,DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D
	};
	//
	uint lignes [BLOQUES] = {0};
	uint decales[BLOQUES] = {0};
	FOR(0, i, BLOQUES) {
		lignes[i] = rand() % EMA_INTS;
		decales[i] = rand() % MAX_DECALES;
	};
	//	Assurances :
	FOR(0, i, EMA_INTS) {
		lignes [i] = i;
		decales[i] = 0;
	}
	//
	Mdl_t * mdl = cree_mdl(Y, insts, lignes, decales);*/


	/*

		De temps en temps. Echanger 2 connections.
	
	#	Juste avant aller_retour(mdl, t0, t1);
	for i in range(QUANTITE aleatoire):
		p[c][i], p[c][j] = p[c][j], p[c][i]

		Sans contre parties. Si l'optimisation aime, elle gardera (et modifira).
	Si elle aime pas elle annulera ce poids ou en fera autre chose.

		C'est un perturbateur qui ralentie mais qui propose des alternatives
	que la descente du gradient ne prendrait pas forcement d'elle meme.

		Ca ralentie certe, mais au moins ca decouvre plus (sans devoire meme de l'aleatoire).

	!! -> C'est comme le DROP_out. Ca se reequilibrera mais ca permet de decouvrire.

	*/

	/*	==============

		Faire un 
		FOR(0, i, POIDS)
			poid[i] += (2*rnd()-1)*0.10
	*/

	Mdl_t * mdl = ouvrire_mdl("mdl.bin");

	plumer_mdl(mdl);

	//	================= Initialisation ==============
	uint t0 = DEPART;
	uint t1 = ROND_MODULO(FIN, (16*16));
	printf("t0=%i t1=%i FIN=%i (t1-t0=%i, %%(16*16)=%i)\n", t0, t1, FIN, t1-t0, (t1-t0)%(16*16));
	//
	//plume_pred(mdl, t0, t1);
	//
	uint REP = 150;
	FOR(0, rep, REP) {
		/*FOR(0, i, 10) {
			printf(" ================== %i/10 ================\n", i);
			optimisation_mini_packet(
				mdl,
				t0, t1, 16*16*100,
				alpha, 1.0,
				RMSPROP, 2000,
				pourcent_masque);
			plume_pred(mdl, t0, t1);
			mdl_gpu_vers_cpu(mdl);
			ecrire_mdl(mdl, "mdl.bin");
		}*/
		//
		optimiser(
			mdl,
			t0, t1,
			alpha, 1.0,
			RMSPROP, 2000,
			//pourcent_masque_nulle);
			pourcent_masque);
		//
		mdl_gpu_vers_cpu(mdl);
		ecrire_mdl(mdl, "mdl.bin");
		plume_pred(mdl, t0, t1);
		printf("===================================================\n");
		printf("==================TERMINE %i/%i=======================\n", rep+1, REP);
		printf("===================================================\n");
	}
	//
	mdl_gpu_vers_cpu(mdl);
	ecrire_mdl(mdl, "mdl.bin");
	liberer_mdl(mdl);

	//	-- Fin --
	liberer_tout();
};