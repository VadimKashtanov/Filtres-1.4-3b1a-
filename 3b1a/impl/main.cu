#include "hip/hip_runtime.h"
#include "main.cuh"

#include "../impl_tmpl/tmpl_etc.cu"

static void plume_pred(Mdl_t * mdl, uint t0, uint t1) {
	float * ancien = mdl_pred(mdl, t0, t1, 3);
	printf("PRED GENERALE = ");
	FOR(0, p, P) printf(" %f%% ", 100*ancien[p]);
	printf("\n");
	free(ancien);
};

float pourcent_masque_nulle[C] = {0};

float * pourcent_masque = de_a(0.10, 0.00, C);

//	! A FAIRE ! :
//		selection (mutation de +/- 1 ligne (de meme source))
//

//	# Un jour reflechire a f(x@p0 + b0) * f(x@p1 + b1) + f(x@p2 + b2)

float * alpha = de_a(2e-4, 2e-4, C);

//	## (x/3) * (x-2)**2                     ##
//	## score(x) + rnd()*abs(score(x))*0.05  ##

PAS_OPTIMISER()
int main(int argc, char ** argv) {
	MSG("S(x) Ajouter un peut d'aléatoire");
	MSG("S(x) Eventuellement faire des prediction plus lointaines");
	//	-- Init --
	srand(0);
	hipSetDevice(0);
	titre(" Charger tout ");   charger_tout();

	//	-- Verification --
	//titre("Verifier MDL");     verif_mdl_1e5();

	//===============
	titre("  Programme Generale  ");
	ecrire_structure_generale("structure_generale.bin");

	uint Y[C] = {
		512,
		512,
		256,
		128,
		64,
		32,
		16,
		8,
		P
	};
	uint insts[C] = {
		FILTRES_PRIXS,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D
	};
	//	Assurances :
	ema_int_t * bloque[BLOQUES];
	uint params[MAX_PARAMS];
	FOR(0, i, BLOQUES) {
		uint source     = rand() % SOURCES;
		uint nature     = 0;//rand() % NATURES;
		uint K_ema      = 1;//rand() % MAX_EMA,
		uint intervalle = 1;//rand() % MAX_INTERVALLE,
		uint decale     = 0;//rand() % MAX_DECALES,
		//
		FOR(0, j, MAX_PARAMS) {
			if (max_param[nature][j]-min_param[nature][j] != 0)
				params[j] = min_param[nature][j] + (rand() % (max_param[nature][j]-min_param[nature][j]));
			else
				params[j] = max_param[nature][j];
		}
		//
		bloque[i] = cree_ligne(
			source,
			nature,
			K_ema,
			intervalle,
			decale,
			params
		);
	}
	//
	Mdl_t * mdl = cree_mdl(Y, insts, bloque);

	//Mdl_t * mdl = ouvrire_mdl("mdl.bin");

	enregistrer_les_lignes_brute(mdl, "lignes_brute.bin");


	/*

		De temps en temps. Echanger 2 connections.
	
	#	Juste avant aller_retour(mdl, t0, t1);
	for i in range(QUANTITE aleatoire):
		p[c][i], p[c][j] = p[c][j], p[c][i]

		Sans contre parties. Si l'optimisation aime, elle gardera (et modifira).
	Si elle aime pas elle annulera ce poids ou en fera autre chose.

		C'est un perturbateur qui ralentie mais qui propose des alternatives
	que la descente du gradient ne prendrait pas forcement d'elle meme.

		Ca ralentie certe, mais au moins ca decouvre plus (sans devoire meme de l'aleatoire).

	!! -> C'est comme le DROP_out. Ca se reequilibrera mais ca permet de decouvrire.

	*/

	plumer_mdl(mdl);

	//	================= Initialisation ==============
	uint t0 = DEPART;
	uint t1 = ROND_MODULO(FIN, (16*16));
	printf("t0=%i t1=%i FIN=%i (t1-t0=%i, %%(16*16)=%i)\n", t0, t1, FIN, t1-t0, (t1-t0)%(16*16));
	//
	plume_pred(mdl, t0, t1);
	//comportement(mdl, t0, t0+16*16);
	//
	uint REP = 150;
	FOR(0, rep, REP) {
		FOR(0, i, 10) {
			printf(" ================== %i/10 ================\n", i);
			optimisation_mini_packet(
				mdl,
				t0, t1, 16*16*1,
				alpha, 1.0,
				RMSPROP, 300,
				pourcent_masque);
			plume_pred(mdl, t0, t1);
			mdl_gpu_vers_cpu(mdl);
			ecrire_mdl(mdl, "mdl.bin");
		}
		//
		/*optimiser(
			mdl,
			t0, t1,
			alpha, 1.0,
			RMSPROP, 2000,
			//pourcent_masque_nulle);
			pourcent_masque);*/
		//
		mdl_gpu_vers_cpu(mdl);
		ecrire_mdl(mdl, "mdl.bin");
		plume_pred(mdl, t0, t1);
		printf("===================================================\n");
		printf("==================TERMINE %i/%i=======================\n", rep+1, REP);
		printf("===================================================\n");
	}
	//
	mdl_gpu_vers_cpu(mdl);
	ecrire_mdl(mdl, "mdl.bin");
	liberer_mdl(mdl);

	//	-- Fin --
	liberer_tout();
};