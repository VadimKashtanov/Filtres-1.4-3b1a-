#include "main.cuh"

#include "../impl_tmpl/tmpl_etc.cu"

static void plume_pred(Mdl_t * mdl, uint t0, uint t1) {
	float * ancien = mdl_pred(mdl, t0, t1, 3);
	printf("PRED GENERALE = ");
	FOR(0, p, P) printf(" %f%% ", 100*ancien[p]);
	printf("\n");
	free(ancien);
};

float pourcent_masque_nulle[C] = {0};

float * pourcent_masque = de_a(0.20, 0.00, C);

float * alpha = de_a(2e-4, 2e-4, C);

PAS_OPTIMISER()
int main(int argc, char ** argv) {
	//	-- Init --
	srand(0);
	hipSetDevice(0);
	titre(" Charger tout ");   charger_tout();

	//===============
	titre("  Programme Generale  ");

	ASSERT(argc == 2);
	Mdl_t * mdl = ouvrire_mdl(argv[1]);

	plumer_mdl(mdl);

	//	================= Initialisation ==============
	uint t0 = DEPART;
	uint t1 = ROND_MODULO(FIN, (16*16));
	printf("t0=%i t1=%i FIN=%i (t1-t0=%i, %%(16*16)=%i)\n", t0, t1, FIN, t1-t0, (t1-t0)%(16*16));
	//
	uint REP = 3;
	FOR(0, rep, REP) {
		optimisation_mini_packet(
			mdl,
			t0, t1, 16*16*1,
			alpha, 1.0,
			RMSPROP, 200,
			pourcent_masque);
		plume_pred(mdl, t0, t1);
		mdl_gpu_vers_cpu(mdl);
		ecrire_mdl(mdl, argv[1]);
		//
		printf("===================================================\n");
		printf("================ TERMINE %i/%i  =================\n", rep+1, REP);
		printf("===================================================\n");
	}
	optimiser(
		mdl,
		t0, t1,
		alpha, 1.0,
		RMSPROP, 500,
		pourcent_masque_nulle);
	//
	float _pred = mdl_pred(mdl, t0, t1, 3)[0];
	ecrire<float>("resultat.bin", &_pred, 1);
	//
	mdl_gpu_vers_cpu(mdl);
	ecrire_mdl(mdl, argv[1]);
	liberer_mdl(mdl);

	//	-- Fin --
	liberer_tout();
};