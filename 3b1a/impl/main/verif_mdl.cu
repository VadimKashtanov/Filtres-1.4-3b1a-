#include "hip/hip_runtime.h"
#include "main.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

/*static float _pourcent_masque_nulle[C] = {0};
static float _alpha[C] = {0.01};

__global__
static void kerd_p1e5(float * p, uint i, float _1E5) {
	p[i] += _1E5;
};

static void p1e5(Mdl_t * mdl, uint c, uint i, float _1E5, uint _MODE) {
	if (_MODE == 0) {
		mdl->p[c][i] += _1E5;
	} else {
		kerd_p1e5<<<1,1>>>(mdl->p__d[c], i, _1E5);
		ATTENDRE_CUDA();
	}
};*/

static void __performance() {
	/*ASSERT(C == 11);
	titre("Performance");
	//
	uint Y[C] = {
		512,
		256,
		256,
		256,
		128,
		64,
		32,
		16,
		8,
		4,
		P
	};
	uint insts[C] = {
		FILTRES_PRIXS,
		LSTM1D,
		LSTM1D,
		LSTM1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D,
		DOT1D
	};
	uint lignes[BLOQUES] = {0};
	FOR(0, i, BLOQUES) lignes[i] = rand() % EMA_INTS;
	Mdl_t * mdl = cree_mdl(Y, insts, lignes);
	plumer_mdl(mdl);
	//
	uint plus_T = 16*16*25;
	//
	mdl_plume_grad(mdl, DEPART, DEPART+plus_T);
	//
	printf("TEMPS MODEL = ");
	MESURER(mdl_aller_retour(mdl, DEPART, DEPART+plus_T, 3));
	//
	liberer_mdl(mdl);*/
};

static void __verif_mdl_1e5() {
	/*ASSERT(C == 5);
	titre("Comparer MODEL 1e-5");
	//
	uint Y[C] = {
		64,
		64,
		32,
		8,
		P
	};
	uint insts[C] = {
		FILTRES_PRIXS,
		LSTM1D,
		LSTM1D,
		LSTM1D,
		DOT1D
	};
	uint lignes[BLOQUES] = {
		0
	};
	Mdl_t * mdl = cree_mdl(Y, insts, lignes);
	plumer_mdl(mdl);
	//
	uint plus_T = 16*16*1;
	//
	uint t0 = DEPART;
	uint t1 = ROND_MODULO(FIN, 16*16);
	//
	//mdl_plume_poids(mdl);
	//
	//comportement(mdl, DEPART, DEPART+16*16);
#define MODE 0 //0,1,2,3
	//
	MESURER(mdl_aller_retour(mdl, DEPART, DEPART+plus_T, MODE));
	//mdl_gpu_vers_cpu(mdl);
	//
	//	1e-5
	//
	mdl_zero_gpu(mdl);
	float _f = mdl_score(mdl, DEPART, DEPART+plus_T, MODE);
	float _1E5 = 1e-5;
	FOR(0, c, C) {
		printf("###############################################################\n");
		printf("                       C = %2.i (%s)    \n", c, nom_inst[mdl->insts[c]]);
		printf("#######################vvvvvvvvvvvvvv##########################\n");
		//
		FOR(0, i, mdl->inst_POIDS[c]) {
			p1e5(mdl, c, i, +_1E5, MODE);
			float grad_1e5 = (mdl_score(mdl, DEPART, DEPART+plus_T, MODE) - _f)/_1E5;
			p1e5(mdl, c, i, -_1E5, MODE);
			//
			float a=grad_1e5, b=mdl->dp[c][i];
			printf("%i| ", i);
			PLUME_CMP(a, b);
			printf("\n");
		}
	};
	printf("  1e5 === df(x)  \n");

	//
	liberer_mdl(mdl);*/
};

void verif_mdl_1e5() {
	__performance();
	__verif_mdl_1e5();
};