#include "hip/hip_runtime.h"
#include "mdl.cuh"

//	Sources
float   prixs[PRIXS] = {};
float   macds[PRIXS] = {};
float volumes[PRIXS] = {};
float   hight[PRIXS] = {};
float     low[PRIXS] = {};

float *          prixs__d = 0x0;
float *          macds__d = 0x0;
float *        volumes__d = 0x0;
float *          hight__d = 0x0;
float *            low__d = 0x0;

float * sources[SOURCES] = {
	prixs, macds, volumes, hight, low
};

float * sources__d[SOURCES] = {
	prixs__d, macds__d, volumes__d, hight__d, low__d
};

void charger_les_prixs() {
	uint __PRIXS;
	FILE * fp;
	//
	fp = fopen("prixs/prixs.bin", "rb");
	ASSERT(fp != 0);
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(prixs, sizeof(float), PRIXS, fp);
	fclose(fp);
	//
	fp = fopen("prixs/volumes.bin", "rb");
	ASSERT(fp != 0);
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(volumes, sizeof(float), PRIXS, fp);
	fclose(fp);
	//
	fp = fopen("prixs/macds.bin", "rb");
	ASSERT(fp != 0);
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(macds, sizeof(float), PRIXS, fp);
	fclose(fp);
	//
	fp = fopen("prixs/hight.bin", "rb");
	ASSERT(fp != 0);
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(hight, sizeof(float), PRIXS, fp);
	fclose(fp);
	//
	fp = fopen("prixs/low.bin", "rb");
	ASSERT(fp != 0);
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(low, sizeof(float), PRIXS, fp);
	fclose(fp);
};

//	===========================================================

void ema_int_calc_ema(ema_int_t * ema_int) {
	//			-- Parametres --
	uint K = ema_int->K_ema;
	float _K = 1.0 / ((float)K);
	//	EMA
	ema_int->ema[0] = sources[ema_int->source][0];
	FOR(1, i, FIN) {
		ema_int->ema[i] = sources[ema_int->source][i]*_K + ema_int->ema[i-1] * (1.0 - _K);
	}
};

//	===========================================================

void nature0__direct(ema_int_t * ema_int) {
	//			-- Parametres --
	//			-- Assertions --
	//	-- Transformation des Parametres --
	//		-- Calcule de la Nature --
	FOR(0, i, FIN) ema_int->brute[i] = ema_int->ema[i];
};

static float ema12[PRIXS], ema26[PRIXS], __macd[PRIXS], ema9_macd[PRIXS];

void nature1__macd(ema_int_t * ema_int) {
	//			-- Parametres --
	uint plus0 = ema_int->params[0];
	//			-- Assertions --
	ASSERT(min_param[MACD][0] <= plus0 && plus0 <= max_param[MACD][0]);
	//	-- Transformation des Parametres --
	float K12 = 1.0/(12.0), K26 = 1.0/(12.0), K9 = 1.0/(9.0);
	//		-- Calcule de la Nature --
	//ema12
	ema12[0] = ema_int->ema[0];
	FOR(1, t, FIN) ema12[t] = ema12[t-1]*(K12) + ema_int->ema[t]*(1-K12);
	//ema26
	ema26[0] = ema_int->ema[0];
	FOR(1, t, FIN) ema26[t] = ema26[t-1]*(K26) + ema_int->ema[t]*(1-K26);
	//__macd
	FOR(0, t, FIN) __macd[t] = ema12[t] - ema26[t];
	//ema9 du __macd
	ema9[0] = __macd[0];
	FOR(1, t, FIN) ema9[t] = ema9[t-1]*(K9) + __macd[t]*(1-K9);
	//MACD
	FOR(0, t, FIN) ema_int->brute[t] = __macd[t] - ema9[t];
};

void nature2__chiffre(ema_int_t * ema_int) {
	//			-- Parametres --
	uint cible = ema_int->params[0];
	//			-- Assertions --
	ASSERT(min_param[CHIFFRE][0] <= cible && cible <= max_param[CHIFFRE][0]);
	//	-- Transformation des Parametres --
	float chiffre = (float)cible;
	//		-- Calcule de la Nature --
	FOR(0, t, FIN) {
		float x = ema_int->ema[t];
		ema_int->brute[t] = 2*(chiffre-MIN2(fabs(x-chiffre*roundf((x+0)/chiffre)), fabs(x-chiffre*roundf((x+chiffre)/chiffre))))/chiffre-1
	}
};

void nature3__dx(ema_int_t * ema_int) {
	//			-- Parametres --
	uint plus0 = ema_int->params[0];
	//			-- Assertions --
	ASSERT(min_param[DX][0] <= plus0 && plus0 <= max_param[DX][0]);
	//	-- Transformation des Parametres --
	//		-- Calcule de la Nature --
	FOR(0, t, plus0+1) ema_int->brute[t] = 0;
	FOR(plus0+1, t, FIN) {
		float x = ema_int->ema[t];
		ema_int->brute[t] = ema_int->ema[t] - ema_int->ema[t-1-plus0];
	}
};

void nature4__dxdx(ema_int_t * ema_int) {
	//			-- Parametres --
	uint plus0 = ema_int->params[0];
	uint ema0  = ema_int->params[0];
	uint plus1 = ema_int->params[0];
	//			-- Assertions --
	ASSERT(min_param[DX][0] <= plus0 && plus0 <= max_param[DX][0]);
	ASSERT(min_param[DX][1] <= ema0  && ema0  <= max_param[DX][1]);
	ASSERT(min_param[DX][2] <= plus1 && plus1 <= max_param[DX][2]);
	//	-- Transformation des Parametres --
	float K = 1 / ((float)ema0);
	//		-- Calcule de la Nature --

	//	dx
	FOR(0, t, plus0+1) ema_int->brute[t] = 0;
	FOR(plus0+1, t, FIN) {
		float x = ema_int->ema[t];
		ema_int->brute[t] = ema_int->ema[t] - ema_int->ema[t-1-plus0];
	}

	//	ema0
	ema_int->ema[0] = ema_int->brute[0];
	FOR(1, t, FIN) ema_int->ema[t] = ema_int->ema[t-1]*(K) + ema_int->brute[t]*(1-K);

	//	dxdx
	FOR(0, t, plus1+1) ema_int->brute[t] = 0;
	FOR(plus1+1, t, FIN) {
		float x = ema_int->ema[t];
		ema_int->brute[t] = ema_int->ema[t] - ema_int->ema[t-1-plus1];
	}
};	//dx(ema(dx(ema(arr, ema0), plus0), ema1), plus1)

nature_f fonctions_nature[NATURES] = {
	nature0__direct,
	nature1__macd,
	nature2__chiffre,
	nature3__dx,
	nature4__dxdx
};

ema_int_t * cree_ligne(uint source, uint nature, uint K_ema, uint intervalle, uint decale, uint params[MAX_PARAMS]) {
	ema_int_t * ret = alloc<ema_int_t>(1);
	//
	ret->source = source;
	ret->nature = nature;
	ret->K_ema  = K_ema;
	ret->intervalle = intervalle;
	ret->decale = decale;
	//
	memcpy(mdl->params, params, sizeof(uint) * MAX_PARAMS);
	//
	ema_int_calc_ema(ret);
	fonctions_nature[nature](ret);
	//
	return ret;
};

void liberer_ligne(ema_int_t * ema_int) {
	CONTROLE_CUDA(hipFree(ema_int->    normalisee__d));
	CONTROLE_CUDA(hipFree(ema_int->dif_normalisee__d));
};

void charger_vram_nvidia() {
	CONTROLE_CUDA(hipMalloc((void**)&  prixs__d, sizeof(float) * PRIXS));
	CONTROLE_CUDA(hipMalloc((void**)&  macds__d, sizeof(float) * PRIXS));
	CONTROLE_CUDA(hipMalloc((void**)&volumes__d, sizeof(float) * PRIXS));
	CONTROLE_CUDA(hipMalloc((void**)&  hight__d, sizeof(float) * PRIXS));
	CONTROLE_CUDA(hipMalloc((void**)&    low__d, sizeof(float) * PRIXS));
	//
	CONTROLE_CUDA(hipMemcpy(  prixs__d,   prixs, sizeof(float) * PRIXS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(  macds__d,   macds, sizeof(float) * PRIXS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(volumes__d, volumes, sizeof(float) * PRIXS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(  hight__d, volumes, sizeof(float) * PRIXS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(    low__d, volumes, sizeof(float) * PRIXS, hipMemcpyHostToDevice));
};

void     liberer_cudamalloc() {
	CONTROLE_CUDA(hipFree(  prixs__d));
	CONTROLE_CUDA(hipFree(  macds__d));
	CONTROLE_CUDA(hipFree(volumes__d));
	CONTROLE_CUDA(hipFree(  hight__d));
	CONTROLE_CUDA(hipFree(    low__d));
};

void charger_tout() {
	printf("charger_les_prixs : ");      MESURER(charger_les_prixs());
	printf("calculer_ema_norm_diff : "); MESURER(calculer_ema_norm_diff());
	printf("charger_les_prixs : ");      MESURER(charger_vram_nvidia());
};

void liberer_tout() {
	titre("Liberer tout");
	liberer_cudamalloc();
};